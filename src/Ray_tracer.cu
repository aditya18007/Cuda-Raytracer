#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdint>
#include <iostream>
#include <climits>
#include "Frame.h"
#include "Camera.h"
#include "glm/geometric.hpp"
#include "Dimensions.h"
#include "Object_Loader.h"
#include "Application.h"
using namespace std;

struct World{
	glm::vec3 bgcolor;
};

struct Material{
	glm::vec3 color;
};

struct Ray {
	glm::vec3 origin;
	glm::vec3 direction;
    float t;
};

__device__ bool intersect_triangle(
		Ray& r, Triangle tri)
{
    const glm::vec3 edge1 = tri.b - tri.a;
    const glm::vec3 edge2 = tri.c - tri.a;
    const glm::vec3 h = glm::cross( r.direction, edge2 );
    const float a = glm::dot( edge1, h );
    if (a > -0.0001f && a < 0.0001f) return false; // ray parallel to triangle
    const float f = 1 / a;
    const glm::vec3 s = r.origin - tri.a;
    const float u = f * glm::dot( s, h );
    if (u < 0 || u > 1) return false;
    const glm::vec3 q = cross( s, edge1 );
    const float v = f * dot( r.direction, q );
    if (v < 0 || u + v > 1) return false;
    float t_poss = f * dot( edge2, q );
    if (r.t > 0.0001f) r.t = min( t_poss, r.t );
    return true;
}


__global__ void ray_trace(hipSurfaceObject_t surface, const glm::vec3 camera_pos, glm::vec3 u, glm::vec3 v, glm::vec3 dir, Triangle* d_triangles, int n_triangles)
{

    //pixel index
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if(x >= WIDTH)
        return;
    if(y >= HEIGHT)
        return;

    float SMALLEST_DIST = 1e-4;

    World world;
    world.bgcolor = glm::vec3(0.28, 0.28, 0.28);


    float xw = 0.0011111111111111111f*x - 0.9994444444444445;
    float yw = (y - HEIGHT/2.0f)/HEIGHT + 0.0005555555555555556;

    dir += u * xw;
    dir += v * yw;

    Ray r{};
    r.origin = camera_pos;
    r.direction = normalize(dir);
    r.t = FLT_MAX;
    Material m{};
    m.color =  glm::vec3(0.1, 0.7, 0.0);

    for(int i = 0; i < n_triangles; i++){
        Triangle triangle = d_triangles[i];
        intersect_triangle(r,triangle);
    }
    if (r.t - FLT_MAX > 1.0f || FLT_MAX - r.t > 1.0f){
        uchar4 pixel = { (uint8_t)(m.color.x*255),
                         (uint8_t)(m.color.y*255),
                         (uint8_t)(m.color.z*255),
                         (uint8_t)(1.0*255)};
        surf2Dwrite(pixel, surface, x * sizeof(uchar4), y);
        return;
    }
    uchar4 pixel = { (uint8_t)(world.bgcolor.x*255),
                     (uint8_t)(world.bgcolor.y*255),
                     (uint8_t)(world.bgcolor.z*255),
                     (uint8_t)(1.0*255)};
    surf2Dwrite(pixel, surface, x * sizeof(uchar4), y);
}

__device__ bool intersect_bbox(Ray& r, float min_x, float min_y, float min_z, float max_x, float max_y, float max_z ){

    float tx1 = (min_x - r.origin.x) / r.direction.x, tx2 = (max_x - r.origin.x) / r.direction.x;
    float tmin = min( tx1, tx2 ), tmax = max( tx1, tx2 );

    float ty1 = (min_y - r.origin.y) / r.direction.y, ty2 = (max_y - r.origin.y) / r.direction.y;
    tmin = max( tmin, min( ty1, ty2 ) ), tmax = min( tmax, max( ty1, ty2 ) );

    float tz1 = (min_z - r.origin.z) / r.direction.z, tz2 = (max_z - r.origin.z) / r.direction.z;
    tmin = max( tmin, min( tz1, tz2 ) ), tmax = min( tmax, max( tz1, tz2 ) );

    return tmax >= tmin && tmin < r.t && tmax > 0;
}

__device__ void IntersectBVH( Ray& ray, Triangle* d_triangles, int* d_triangle_indices, BVH_node* tree, const uint nodeIdx )
{
    BVH_node& node = tree[nodeIdx];
    if (!intersect_bbox( ray, node.min_x, node.min_y, node.min_z, node.max_x, node.max_y, node.max_z )) return;
    if (node.is_leaf()) {
        for (uint i = 0; i < node.prim_count; i++ ){
            auto& tri = d_triangles[ d_triangle_indices[node.start_idx + i] ];
            intersect_triangle(ray, tri);
        }
    } else {
        IntersectBVH( ray, d_triangles, d_triangle_indices, tree, node.left_node );
        IntersectBVH( ray, d_triangles, d_triangle_indices, tree, node.left_node+1 );
    }
}

__global__ void ray_trace2(hipSurfaceObject_t surface, const glm::vec3 camera_pos, glm::vec3 u, glm::vec3 v, glm::vec3 dir, Triangle* d_triangles, int* d_triangle_indices, int n_triangles, BVH_node* d_traversal_tree, int d_traversal_tree_size)
{

    //pixel index
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if(x >= WIDTH)
        return;
    if(y >= HEIGHT)
        return;

    float SMALLEST_DIST = 1e-4;

    World world;
    world.bgcolor = glm::vec3(0.28, 0.28, 0.28);


    float xw = 0.0011111111111111111f*x - 0.9994444444444445;
    float yw = (y - HEIGHT/2.0f)/HEIGHT + 0.0005555555555555556;

    dir += u * xw;
    dir += v * yw;

    Ray r{};
    r.origin = camera_pos;
    r.direction = normalize(dir);
    r.t = FLT_MAX;
    Material m{};
    m.color =  glm::vec3(0.1, 0.7, 0.0);

//    for(int i = 0; i < n_triangles; i++){
//        Triangle triangle = d_triangles[i];
//        intersect_triangle(r,triangle);
//    }
    IntersectBVH(r, d_triangles, d_triangle_indices, d_traversal_tree, 0);
    if (r.t - FLT_MAX > 1.0f || FLT_MAX - r.t > 1.0f){
        uchar4 pixel = { (uint8_t)(m.color.x*255),
                         (uint8_t)(m.color.y*255),
                         (uint8_t)(m.color.z*255),
                         (uint8_t)(1.0*255)};
        surf2Dwrite(pixel, surface, x * sizeof(uchar4), y);
        return;
    }
    uchar4 pixel = { (uint8_t)(world.bgcolor.x*255),
                     (uint8_t)(world.bgcolor.y*255),
                     (uint8_t)(world.bgcolor.z*255),
                     (uint8_t)(1.0*255)};
    surf2Dwrite(pixel, surface, x * sizeof(uchar4), y);
}

extern void compute_frame(Frame& frame, const Camera& camera, Triangle* d_triangles, int n_triangles){
    auto surface = frame.get_bitmap_surface();

    int num_threads_x = 32;
    int num_threads_y = 32;
    dim3 block_shape = dim3( num_threads_x, num_threads_y ,1);


    int num_blocks_x = ( WIDTH / num_threads_x) + 1;
    int num_blocks_y = ( HEIGHT / num_threads_y) + 1;

    dim3 grid_shape = dim3( num_blocks_x, num_blocks_y , 1);

    ray_trace<<<grid_shape, block_shape>>>(surface, camera.get_camera_position(), camera.get_u(), camera.get_v(), camera.get_dir(), d_triangles, n_triangles);
    hipDeviceSynchronize();
}

extern void compute_frame(Frame& frame, const Camera& camera, Triangle* d_triangles, int* d_triangle_indices, int n_triangles, BVH_node* d_traversal_tree, int d_traversal_tree_size){
    auto surface = frame.get_bitmap_surface();

    int num_threads_x = 32;
    int num_threads_y = 32;
    dim3 block_shape = dim3( num_threads_x, num_threads_y ,1);


    int num_blocks_x = ( WIDTH / num_threads_x) + 1;
    int num_blocks_y = ( HEIGHT / num_threads_y) + 1;

    dim3 grid_shape = dim3( num_blocks_x, num_blocks_y , 1);

    ray_trace2<<<grid_shape, block_shape>>>(surface, camera.get_camera_position(), camera.get_u(), camera.get_v(), camera.get_dir(), d_triangles, d_triangle_indices,n_triangles, d_traversal_tree, d_traversal_tree_size);
    hipDeviceSynchronize();
}